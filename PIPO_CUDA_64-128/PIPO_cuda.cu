
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define ROUND 13
#define SIZE 2
#define MASTER_KEY_SIZE 2
#define BLOCK_CNT 32
#define THREAD_SIZE 32

typedef unsigned char u8;
typedef unsigned int u32;


u32 MASTER_KEY[MASTER_KEY_SIZE * SIZE * BLOCK_CNT] = {0, };
u32 ROUND_KEY[(ROUND+1) * SIZE];
u32 PLAIN_TEXT[SIZE * BLOCK_CNT] = {0, };
u32 CIPHER_TEXT[SIZE * BLOCK_CNT] = {0, };

__device__ void key_add(u8* X, u8* rk)
{
	for(int i=0;i<8;i++)
		X[i] ^= rk[i];
}

__device__ void S_layer(u8 *X)
{
    u8 T[3] = { 0, };
    //(MSB: x[7], LSB: x[0]) 
    // Input: x[7], x[6], x[5], x[4], x[3], x[2], x[1], x[0] 
    //S5_1
    X[5] ^= (X[7] & X[6]);
    X[4] ^= (X[3] & X[5]);
    X[7] ^= X[4];
    X[6] ^= X[3];
    X[3] ^= (X[4] | X[5]);
    X[5] ^= X[7];
    X[4] ^= (X[5] & X[6]);
    //S3
    X[2] ^= X[1] & X[0];
    X[0] ^= X[2] | X[1];
    X[1] ^= X[2] | X[0];
    X[2] = ~X[2];
    // Extend XOR
    X[7] ^= X[1];   X[3] ^= X[2];   X[4] ^= X[0];
    //S5_2
    T[0] = X[7];    T[1] = X[3];    T[2] = X[4];
    X[6] ^= (T[0] & X[5]);
    T[0] ^= X[6];
    X[6] ^= (T[2] | T[1]);
    T[1] ^= X[5];
    X[5] ^= (X[6] | T[2]);
    T[2] ^= (T[1] & T[0]);
    // Truncate XOR and bit change
    X[2] ^= T[0];   T[0] = X[1] ^ T[2]; X[1] = X[0]^T[1];   X[0] = X[7];    X[7] = T[0];
    T[1] = X[3];    X[3] = X[6];    X[6] = T[1];
    T[2] = X[4];    X[4] = X[5];    X[5] = T[2];
    // Output: (MSb) x[7], x[6], x[5], x[4], x[3], x[2], x[1], x[0] (LSb)
}

__device__ void R_layer(u8* X)
{
    X[1] = ((X[1] << 7)) | ((X[1] >> 1));
    X[2] = ((X[2] << 4)) | ((X[2] >> 4));
    X[3] = ((X[3] << 3)) | ((X[3] >> 5));
    X[4] = ((X[4] << 6)) | ((X[4] >> 2));
    X[5] = ((X[5] << 5)) | ((X[5] >> 3));
    X[6] = ((X[6] << 1)) | ((X[6] >> 7));
    X[7] = ((X[7] << 2)) | ((X[7] >> 6));
}

__device__ void inv_R_layer(u8* X)
{
	X[1] = ((X[1] << 1)) | ((X[1] >> 7));
	X[2] = ((X[2] << 4)) | ((X[2] >> 4));
	X[3] = ((X[3] << 5)) | ((X[3] >> 3));
	X[4] = ((X[4] << 2)) | ((X[4] >> 6));
	X[5] = ((X[5] << 3)) | ((X[5] >> 5));
	X[6] = ((X[6] << 7)) | ((X[6] >> 1));
	X[7] = ((X[7] << 6)) | ((X[7] >> 2));
}

__device__ void inv_S_layer(u8 *X)
{	//(MSB: x[7], LSB: x[0]) 
	// Input: x[7], x[6], x[5], x[4], x[3], x[2], x[1], x[0] 

	u8 T[3] = { 0, };

	T[0] = X[7]; X[7] = X[0]; X[0] = X[1]; X[1] = T[0];
	T[0] = X[7];	T[1] = X[6];	T[2] = X[5];
	// S52 inv
	X[4] ^= (X[3] | T[2]);
	X[3] ^= (T[2] | T[1]);
	T[1] ^= X[4];
	T[0] ^= X[3];
	T[2] ^= (T[1] & T[0]);
	X[3] ^= (X[4] & X[7]);
	//  Extended XOR
	X[0] ^= T[1]; X[1] ^= T[2]; X[2] ^= T[0];	
	T[0] = X[3]; X[3] = X[6]; X[6] = T[0];
	T[0] = X[5]; X[5] = X[4]; X[4] = T[0];
	//  Truncated XOR
	X[7] ^= X[1];	X[3] ^= X[2];	X[4] ^= X[0];
	// Inv_S5_1
	X[4] ^= (X[5] & X[6]);
	X[5] ^= X[7];
	X[3] ^= (X[4] | X[5]);
	X[6] ^= X[3];
	X[7] ^= X[4];
	X[4] ^= (X[3] & X[5]);
	X[5] ^= (X[7] & X[6]);
	// Inv_S3
	X[2] = ~X[2];
	X[1] ^= X[2] | X[0];
	X[0] ^= X[2] | X[1];
	X[2] ^= X[1] & X[0];
	 // Output: x[7], x[6], x[5], x[4], x[3], x[2], x[1], x[0]
}


__global__ void encrypt(u32* PLAIN_TEXT, u32* ROUND_KEY)
{
	int i=0;
	__shared__ u32 shared_RK[(ROUND+1) * SIZE];
	if(threadIdx.x <= (ROUND+1) * SIZE)
	{
		shared_RK[threadIdx.x] = ROUND_KEY[threadIdx.x];
	}	
	__syncthreads();

	u8* P = (u8*)(PLAIN_TEXT + (blockIdx.x * blockDim.x + threadIdx.x) * 2);
	u8* RK = (u8*)shared_RK;

	key_add(P, RK);
	printf("BEFORE\n");
	printf("%02x%02x%02x%02x %02x%02x%02x%02x\n", P[7], P[6], P[5], P[4], P[3], P[2], P[1], P[0]);
	for(i=1;i<=ROUND;i++)
	{
		S_layer(P);
		R_layer(P);
		key_add(P, RK + (i * 8));
	}
	printf("AFTER\n");
	printf("%02x%02x%02x%02x %02x%02x%02x%02x\n\n", P[7], P[6], P[5], P[4], P[3], P[2], P[1], P[0]);
}

__global__ void decrypt(u32* CIPHER_TEXT, u32* ROUND_KEY)
{
	int i=0;
	__shared__ u32 shared_RK[(ROUND+1) * SIZE];
	if(threadIdx.x <= (ROUND+1) * SIZE)
	{
		shared_RK[threadIdx.x] = ROUND_KEY[threadIdx.x];
	}	
	__syncthreads();

	u8* C = (u8*)(CIPHER_TEXT + (blockIdx.x * blockDim.x + threadIdx.x) * 2);
	u8* RK = (u8*)shared_RK;

	printf("BEFORE\n");
	printf("%02x%02x%02x%02x, %02x%02x%02x%02x\n", C[7], C[6], C[5], C[4], C[3], C[2], C[1], C[0]);
	for(i=ROUND;i>0;i--)
	{
		key_add(C, RK + (i * 8));
		inv_R_layer(C);
		inv_S_layer(C);
	}
	key_add(C, RK);
	printf("AFTER\n");
	printf("%02x%02x%02x%02x, %02x%02x%02x%02x\n\n", C[7], C[6], C[5], C[4], C[3], C[2], C[1], C[0]);
}
/*
void getGapTime(struct timeval* start_time, struct timeval* end_time, struct timeval* gap_time)
{
	gap_time->tv_sec = end_time->tv_sec - start_time->tv_sec;
	gap_time->tv_usec = end_time->tv_usec - start_time->tv_usec;
	if(gap_time->tv_usec < 0)
 	{
		gap_time->tv_usec = gap_time->tv_usec + 1000000;
		gap_time->tv_sec -= 1;
	}
}

float timevalToFloat(struct timeval* time)
{
	double val;
	val = time->tv_sec;
	val += (time->tv_usec * 0.000001);
	return val;
}
*/
int main()
{
	u32 i, j, k;
	u32 RCON;

    // master key test vector
    MASTER_KEY[0] = 0x2E152297;
    MASTER_KEY[1] = 0x7E1D20AD;
    MASTER_KEY[2] = 0x779428D2;
    MASTER_KEY[3] = 0x6DC416DD;
	// generate round key
    RCON = 0;
	for(i=0;i<=ROUND;i++)
	{
		for(j=0;j<SIZE;j++)
			ROUND_KEY[i * SIZE + j] = MASTER_KEY[(SIZE * i + j) % (MASTER_KEY_SIZE * SIZE)];
		ROUND_KEY[SIZE * i] ^= RCON;
		RCON++;
	}
    for(k=0;k<BLOCK_CNT;k++)
	{
		// Plain text test vector
		PLAIN_TEXT[k * 2] = 0x1E270026;
		PLAIN_TEXT[k * 2 + 1] = 0x098552F6;
	}
    // encrypt

	u32 *dev_PLAIN_TEXT, *dev_ROUND_KEY;
	hipMalloc((void**)&dev_PLAIN_TEXT, SIZE * BLOCK_CNT * sizeof(u32));
	hipMalloc((void**)&dev_ROUND_KEY, (ROUND+1) * SIZE * sizeof(u32));

	hipMemcpy(dev_PLAIN_TEXT, PLAIN_TEXT, SIZE * BLOCK_CNT * sizeof(u32), hipMemcpyHostToDevice);
	hipMemcpy(dev_ROUND_KEY, ROUND_KEY, (ROUND+1) * SIZE * sizeof(u32), hipMemcpyHostToDevice);

    //struct timeval gpu_start, gpu_end, gap;
    //gettimeofday(&gpu_start, NULL);
	encrypt<<<BLOCK_CNT/THREAD_SIZE, THREAD_SIZE>>>(dev_PLAIN_TEXT, dev_ROUND_KEY);
	hipDeviceSynchronize();
    //gettimeofday(&gpu_end, NULL);
    //getGapTime(&gpu_start, &gpu_end, &gap);
    //float f_gpu_gap = timevalToFloat(&gap);
    //printf("%.6f\n", f_gpu_gap);

	hipMemcpy(CIPHER_TEXT, dev_PLAIN_TEXT, SIZE * BLOCK_CNT * sizeof(u32), hipMemcpyDeviceToHost);

	hipFree(dev_PLAIN_TEXT);
	hipFree(dev_ROUND_KEY);
 
    /*
    for(i=0;i<BLOCK_CNT;i++)
    {
        printf("%08x %08x\n", CIPHER_TEXT[i * 2 + 1], CIPHER_TEXT[i * 2]);
    }
    
    u32 *dev_CIPHER_TEXT;
    cudaMalloc((void**)&dev_CIPHER_TEXT, SIZE * BLOCK_CNT * sizeof(u32));
    cudaMemcpy(dev_CIPHER_TEXT, CIPHER_TEXT, SIZE * BLOCK_CNT * sizeof(u32), cudaMemcpyHostToDevice);
    cudaMemcpy(dev_ROUND_KEY, ROUND_KEY, (ROUND+1) * SIZE * sizeof(u32), cudaMemcpyHostToDevice);
    decrypt<<<1, BLOCK_CNT>>>(dev_CIPHER_TEXT, dev_ROUND_KEY);
    cudaDeviceSynchronize();
    cudaMemcpy(PLAIN_TEXT, dev_CIPHER_TEXT, SIZE * BLOCK_CNT * sizeof(u32), cudaMemcpyDeviceToHost);

    cudaFree(dev_CIPHER_TEXT);
    cudaFree(dev_ROUND_KEY);

    for(i=0;i<BLOCK_CNT;i++)
    {
        printf("%08x %08x\n", PLAIN_TEXT[i * 2 + 1], PLAIN_TEXT[i * 2]);
    }
    */

	return 0;
}